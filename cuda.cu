#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define row 10000
#define col 10000

int matrixA[row][col], matrixB[row][col], matrixC[row][col], matrixD[row][col];

__global__ void add_matrix(int matrixA[row][col], int matrixB[row][col], int matrixC[row][col])
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;

	if (i < row && j < col)
	{
		matrixC[i][j] = matrixA[i][j] + matrixB[i][j];
	}
}

int main()
{
	
	int(*deviceA)[col];
	int(*deviceB)[col];
	int(*deviceC)[col];
	int i, j;
	for (i = 0; i < row; i++)
	{
		for (j = 0; j < col; j++)
		{
			matrixA[i][j] = rand() % 100;
			matrixB[i][j] = rand() % 100;
		}
	}
	/*for (i = 0; i < row; i++)
	{
	for (j = 0; j < col; j++)
	{
	printf("%d ", matrixA[i][j]);
	}
	printf("\n");
	}*/
	hipEvent_t start_time, stop_time;
	float elapsedTime;
	

	//clock_t start_time = clock();
	hipMalloc((void **)&deviceA, row * col * sizeof(int));
	hipMalloc((void **)&deviceB, row * col * sizeof(int));
	hipMalloc((void **)&deviceC, row * col * sizeof(int));

	hipMemcpy(deviceA, matrixA, row * col * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceB, matrixB, row * col * sizeof(int), hipMemcpyHostToDevice);

	dim3 threadsPerBlock(32, 32);
	dim3 numOfBlocks(ceil(row / 32.0), ceil(col / 32.0));
	hipEventCreate(&start_time);
	hipEventRecord(start_time, 0);

	add_matrix <<<numOfBlocks, threadsPerBlock >>> (deviceA, deviceB, deviceC);
	
	hipEventCreate(&stop_time);
	hipEventRecord(stop_time, 0);
	hipEventSynchronize(stop_time);

	hipEventElapsedTime(&elapsedTime, start_time, stop_time);
	hipMemcpy(matrixC, deviceC, row*col * sizeof(int), hipMemcpyDeviceToHost);

	//clock_t stop_time = clock();
	
	printf(" Parallely Elapsed Time: %f ms\n", elapsedTime);

	
	clock_t start_time_nonparallely, stop_time_nonparallely;
	start_time_nonparallely = clock();

	for (i = 0; i < row; i++)
	{
		for (j = 0; j < col; j++)
		{
			matrixD[i][j] = matrixA[i][j] + matrixB[i][j];
		}
	}
	stop_time_nonparallely = clock();
	printf("Non-parallely Elapsed Time: %f ms\n", (float)((stop_time_nonparallely) - (start_time_nonparallely)));
	
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);
}